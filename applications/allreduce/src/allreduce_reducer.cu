#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES, ALL RIGHTS RESERVED.
 *
 * This software product is a proprietary product of NVIDIA CORPORATION &
 * AFFILIATES (the "Company") and all right, title, and interest in and to the
 * software product, including all associated intellectual property rights, are
 * and shall remain exclusively with the Company.
 *
 * This software product is governed by the End User License Agreement
 * provided with the software product.
 *
 */
#include <hip/hip_runtime_api.h>
#include <list>

#include <doca_log.h>

#include "allreduce_reducer.h"

#define CEIL_DEV(n, divisor) (((n) + (divisor) - 1) / (divisor)) /* Round-up integers division */
#define MIN_VEC_LEN_PER_THREAD 48 /* Should always be above 32 for best performance \
				   * Using 48 because of rounding up errors */

DOCA_LOG_REGISTER(ALLREDUCE::Reducer::GPU);

/*
 * Function that gets dst and src vector, start and end indices, step size then produces
 * the vectors into dst from "start" (inclusive) to "end" (exclusive) every "step"-th index
 */
typedef void (*prod_func)(void *, void *, size_t, size_t, size_t);

struct vectors {
	union {
		void **arr;	/* if n > 1, pointer to GPU memory containing pointers to n vectors (in GPU memory) */
		void *vec;	/* if n is 1, pointer to GPU memory containing the single vector */
	};
	size_t n;	/* Number of vectors in struct, determines union */
};

static struct {
	int dev_id;			/* GPU device ID */
	int max_thrds_per_blk;		/* Maximum supported number of threads in a single block */
	int max_grid_dim_x;		/* Maximum X dimension of a kernel grid */
	int max_grid_dim_y;		/* Maximum Y dimension of a kernel grid */
	int warp_size;			/* Number of parallel processing units in a single GPU multiprocessor */
} gpu_info;

__constant__ enum allreduce_operation operation;
__constant__ enum allreduce_datatype datatype;

/*
 * Sums the vectors into dst_vector from "from" (inclusive) to "to" (exclusive) every "step"-th index
 *
 * @dst_vector [in]: CUDA memory that holds an array of numbers
 * @src_vector [in]: CUDA memory that holds an array of numbers
 * @from [in]: Index to start from the summation process (inclusive)
 * @to [in]: Index to stop the summation process when reached (exclusive)
 * @step [in]: The step between two consecutive indexes
 */
__device__ static void
gpu_summation(void *dst_vector, void *src_vector, size_t from, size_t to, size_t step)
{
	size_t i = from;

	switch (datatype) {
	case ALLREDUCE_BYTE:
		for (; i < to; i += step)
			((uint8_t *)dst_vector)[i] += ((uint8_t *)src_vector)[i];
		break;
	case ALLREDUCE_INT:
		for (; i < to; i += step)
			((int *)dst_vector)[i] += ((int *)src_vector)[i];
		break;
	case ALLREDUCE_FLOAT:
		for (; i < to; i += step)
			((float *)dst_vector)[i] += ((float *)src_vector)[i];
		break;
	case ALLREDUCE_DOUBLE:
		for (; i < to; i += step)
			((double *)dst_vector)[i] += ((double *)src_vector)[i];
		break;
	}
}

/*
 * Multiply the vectors element-element into dst_vector from "from" (inclusive) to "to" (exclusive) every "step"-th index
 *
 * @dst_vector [in]: CUDA memory that holds an array of numbers
 * @src_vector [in]: CUDA memory that holds an array of numbers
 * @from [in]: Index to start from the product process (inclusive)
 * @to [in]: Index to stop the product process when reached (exclusive)
 * @step [in]: The step between two consecutive indexes
 */
__device__ static void
gpu_product(void *dst_vector, void *src_vector, size_t from, size_t to, size_t step)
{
	size_t i = from;

	switch (datatype) {
	case ALLREDUCE_BYTE:
		for (; i < to; i += step)
			((uint8_t *)dst_vector)[i] *= ((uint8_t *)src_vector)[i];
		break;
	case ALLREDUCE_INT:
		for (; i < to; i += step)
			((int *)dst_vector)[i] *= ((int *)src_vector)[i];
		break;
	case ALLREDUCE_FLOAT:
		for (; i < to; i += step)
			((float *)dst_vector)[i] *= ((float *)src_vector)[i];
		break;
	case ALLREDUCE_DOUBLE:
		for (; i < to; i += step)
			((double *)dst_vector)[i] *= ((double *)src_vector)[i];
		break;
	}
}

/*
 * Iterativly reduces the vectors with dst_vec on the GPU.
 * Every CUDA block operates on all vectors, but only in a specific index range
 *
 * Complexity: O( m * max(MIN_VEC_LEN_PER_THREAD, n/(B*T)) ) where n is vec_len, m is nb_vecs,
 * B is the number of blocks in the grid, T is the number of threads per block.
 *
 * @dst_vec [in]: CUDA memory that holds an array of numbers
 * @src [in]: Holds 1 or more vectors to be reduced with the dst_vec
 * @vec_len [in]: The length of all the vectors, in "datatype" units
 */
__global__ static void
_gpu_reduce(void *dst_vec, struct vectors src, size_t vec_len)
{
	size_t start, block_sub_vec_len, end;
	prod_func gpu_apply;
	size_t i;

	/* Choose reduce process */
	switch (operation) {
	case ALLREDUCE_SUM:
		gpu_apply = gpu_summation;
		break;
	case ALLREDUCE_PROD:
		gpu_apply = gpu_product;
		break;
	default:
		/* Can never happen, initialization check this value is a valid enum */
		return;
	}

	/* Calculate the number of elements each block reduces */
	block_sub_vec_len = CEIL_DEV(vec_len, gridDim.x * gridDim.y);
	/* Calculate the offset of the sub vector this block should work on */
	start = (blockIdx.x * gridDim.y + blockIdx.y) * block_sub_vec_len;
	end = start + block_sub_vec_len;
	/* Offset block start to thread start */
	start += threadIdx.x;
	if (start >= vec_len)
		return;  /* Can happen only in the 2D dim case and for very small vectors */
	/* True only for the last block, set end boundary with regard to real length */
	if (end > vec_len)
		end = vec_len;

	/* Invoke reduce process */
	if (src.n > 1)
		for (i = 0; i < src.n; ++i) {
			gpu_apply(dst_vec, src.arr[i], start, end, blockDim.x);
		}
	else
		gpu_apply(dst_vec, src.vec, start, end, blockDim.x);
}

/*
 * Launches a CUDA kernel that iterativly reduces the vectors with dst_vec on the GPU.
 * Every CUDA block operates on all vectors, but only in a specific index range
 *
 * Complexity: O( m * max(MIN_VEC_LEN_PER_THREAD, n/(B*T)) ) where n is vec_len, m is nb_vecs,
 * B is the number of blocks in the grid, T is the number of threads per block.
 *
 * @dst_vec [in]: CUDA memory that holds an array of numbers
 * @src [in]: Holds 1 or more vectors to be reduced with the dst_vec
 * @vec_len [in]: The length of all the vectors, in "datatype" units
 * @stream [in]: Launch the kernel with this CUDA stream
 */
static inline void
gpu_reduce(void *dst_vec, struct vectors src, size_t vec_len, hipStream_t stream)
{
	dim3 dim;
	size_t nb_warps;
	size_t opt_nb_blks;
	const int recommended_nb_of_warps = 4;

	/* Choosing the parameters for the GPU */
	opt_nb_blks = CEIL_DEV(vec_len, MIN_VEC_LEN_PER_THREAD * gpu_info.warp_size * recommended_nb_of_warps);
	if (opt_nb_blks <= gpu_info.max_grid_dim_x) {
		dim.x = opt_nb_blks;
		nb_warps = recommended_nb_of_warps;
	} else if (opt_nb_blks <= gpu_info.max_grid_dim_x * gpu_info.max_grid_dim_y) {
		/* Prefer division by the lower limit, to problisticly minimaize the number of extra blocks
		 * that are out of range */
		dim.y = gpu_info.max_grid_dim_y;
		dim.x = CEIL_DEV(vec_len, gpu_info.max_grid_dim_y * MIN_VEC_LEN_PER_THREAD * gpu_info.warp_size *
						  recommended_nb_of_warps);
		nb_warps = recommended_nb_of_warps;
	} else {
		dim.x = gpu_info.max_grid_dim_x;
		dim.y = gpu_info.max_grid_dim_y;
		if (vec_len < dim.x * dim.y * gpu_info.max_thrds_per_blk * MIN_VEC_LEN_PER_THREAD) {
			/* "vec_len / (dim.x * dim.y * MIN_VEC_LEN_PER_THREAD)" is the optimal number of threads if
			 * warp size wasn't a concern. So we choose the largest number of threads that is divisible by
			 * warp_size (effictivly increasing the work of each thread to above MIN_VEC_LEN_PER_THREAD) */
			nb_warps = CEIL_DEV(vec_len, dim.x * dim.y * MIN_VEC_LEN_PER_THREAD) / gpu_info.warp_size;
		}
		else
			nb_warps = gpu_info.max_thrds_per_blk / gpu_info.warp_size;
	}

	/* Launching CUDA kernel */
	_gpu_reduce<<<dim, nb_warps * gpu_info.warp_size, 0, stream>>>(dst_vec, src, vec_len);
}

/*
 * Reduces all the vectors will dst_vec, saving the result into dst_vec.
 * Every block operates on all vectors in the given range.
 * Reduction is performed in iterations, where each thread reduce two vectors leaving only half of the vectors to the
 * next iteration.
 *
 * Complexity: O( n * log(m) ) where n is "end - start" and m is the number of vectors.
 *
 * @dst_vec [in]: CUDA memory that holds an array of numbers
 * @vectors [in]: Vectors to be reduced with the dst_vec
 * @start [in]: Index to start from the reduce process (inclusive)
 * @end [in]: Index to stop the reduce process when reached (exclusive)
 *
 * @NOTE: "blockDim.x" MUST be equal to half the number of vectors (excluding dst_vec)
 * @NOTE: all vectors are modified by this functions
 */
__device__ static void
_reduce_all_for_many_vecs_algo(void *dst_vec, void **vectors, size_t start, size_t end)
{
	size_t lvec_idx = 2 * threadIdx.x;
	size_t rvec_idx = lvec_idx + 1;
	size_t min_rvec_idx = 1;  /* The minimum rvec_idx a threadblock holds */
	prod_func gpu_apply;

	switch (operation)
	{
	case ALLREDUCE_SUM:
		gpu_apply = gpu_summation;
		break;
	case ALLREDUCE_PROD:
		gpu_apply = gpu_product;
		break;
	default:
		/* Can never happen, initialization check this value is a valid enum */
		break;
	}

	/* Sum the array - correctness of the algorithm is proven with induction */
	while (min_rvec_idx < (2 * blockDim.x + 1)) {
		if (rvec_idx < (2 * blockDim.x + 1)) {
			gpu_apply(vectors[lvec_idx], vectors[rvec_idx], start, end, 1);
			lvec_idx *= 2;
			rvec_idx *= 2;
		}
		min_rvec_idx *= 2;
		__syncthreads();
	}

	/* Add result to dst */
	start += threadIdx.x;
	if (start < end)
		gpu_apply(dst_vec, vectors[0], start, end, blockDim.x);
}

/*
 * Reduces all the vectors will dst_vec, saving the result into dst_vec.
 * Every block operates on all vectors, but only in a specific index range.
 * All threads in the same block performs reduction in iterations, where each thread reduce two vectors leaving
 * only half of the vectors to the next iteration. When the result is computed - the block continue to process
 * the next "2 * blockDim.x + 1" vectors.
 *
 * Complexity: O( m/T * log(T) * max(MIN_VEC_LEN_PER_THREAD, n/B) )
 * 	       where n is vec_len, m is nb_vecs, B is the number of blocks in the grid, T is min(m, 2 * blockDim.x + 1)
 *
 * @dst_vec [in]: CUDA memory that holds an array of numbers
 * @vectors [in]: Vectors to be reduced with the dst_vec
 * @nb_vecs [in]: The number of vectors in "vectors"
 * @vec_len [in]: The length of all the vectors, in "datatype" units
 *
 * @NOTE: Number of vectors MUST be a whole multiplication of "2 * blockDim.x + 1" (excluding dst_vec)
 * @NOTE: all vectors are modified by this functions
 */
__global__ static void
_reduce_all_for_many_vecs(void *dst_vec, void **vectors, size_t nb_vecs, size_t vec_len)
{
	size_t per_thread_seg_len = CEIL_DEV(vec_len, gridDim.x * gridDim.y);
	size_t start = blockIdx.x * gridDim.y * per_thread_seg_len + blockIdx.y * per_thread_seg_len;
	size_t end = start + per_thread_seg_len;

	if (start >= vec_len)
		return;  /* Can happen only in the 2D dim case */
	if (end > vec_len)
		end = vec_len;

	/* Each iteration sums 2*blockDim.x+1 vectors and the dst_vector. */
	void **end_vecs = vectors + nb_vecs - nb_vecs % (2 * blockDim.x + 1);
	do {
		_reduce_all_for_many_vecs_algo(dst_vec, vectors, start, end);
		vectors += (2 * blockDim.x + 1);
	} while (vectors < end_vecs);
}

/*
 * Reduces all the vectors will dst_vec, saving the result into dst_vec. Using a specific algorithm that takes
 * advantage of the parallelism of the GPU to reduce multiple vectors at the same time.
 *
 * Complexity: O( m/T * log(T) * max(MIN_VEC_LEN_PER_THREAD, n/B) )
 * 	       where n is vec_len, m is nb_vecs, B is the maximum possible number of X blocks in a grid,
 * 	       and T is min(nb_vecs, 2 * "maximum possible number of threads in a block" + 1)
 *
 * @dst_vec [in]: CUDA memory that holds an array of numbers
 * @vectors [in]: Vectors to be reduced with the dst_vec
 * @nb_vecs [in]: The number of vectors in "vectors"
 * @vec_len [in]: The length of all the vectors, in "datatype" units
 * @stream [in]: CUDA stream for async launch of the GPU kernel
 *
 * @NOTE: all vectors are modified by this functions
 */
static void
reduce_all_for_many_vecs(void *dst_vec, void **vectors, size_t nb_vecs, size_t vec_len, hipStream_t stream)
{
	dim3 dim;
	size_t nb_leftover_vecs, opt_nb_blks;

	/* Choosing the parameters for the GPU */
	opt_nb_blks = CEIL_DEV(vec_len, MIN_VEC_LEN_PER_THREAD);
	if (opt_nb_blks <= gpu_info.max_grid_dim_x) {
		dim.x = opt_nb_blks;
	} else {
		/* Prefer division by the lower limit, to problisticly minimaize the number of extra blocks
		 * that are out of range */
		dim.y = gpu_info.max_grid_dim_y;
		opt_nb_blks = CEIL_DEV(vec_len, MIN_VEC_LEN_PER_THREAD * gpu_info.max_grid_dim_y);
		dim.x = (opt_nb_blks <= gpu_info.max_grid_dim_x) ? opt_nb_blks : gpu_info.max_grid_dim_x;
	}

	/* Launching CUDA kernel */
	/* If cannot reduce all vectors using a single kernel, launch max threads to reduce as much as can */
	if (ucs_unlikely(nb_vecs > 2 * gpu_info.max_thrds_per_blk + 1)) {
		/* Can reduce up to 2*gpu_info.max_thrds_per_blk+1 vectors using a single call to the function */
		nb_leftover_vecs = nb_vecs % (2 * gpu_info.max_thrds_per_blk + 1);
		_reduce_all_for_many_vecs<<<dim, gpu_info.max_thrds_per_blk, 0, stream>>>(
			dst_vec, vectors, nb_vecs - nb_leftover_vecs, vec_len);
	} else
		nb_leftover_vecs = nb_vecs;
	/* Reduce all remaining vectors */
	if (nb_leftover_vecs > 1) {
		_reduce_all_for_many_vecs<<<dim, nb_leftover_vecs / 2, 0, stream>>>(
			dst_vec, vectors + (nb_vecs - nb_leftover_vecs), nb_leftover_vecs, vec_len);
	} else if (nb_leftover_vecs == 1) {
		struct vectors src = {};
		src.vec = vectors[nb_vecs - 1];
		src.n = 1;
		gpu_reduce(dst_vec, src, vec_len, stream);
	}
}

/***** Exported C functions *****/

void
allreduce_reduce_all(struct allreduce_super_request *allreduce_super_request, bool is_peers)
{
	void *dst_vec;
	void **src_vecs;
	size_t nb_vecs;
	size_t vec_len = allreduce_super_request->result_vector_size;

	if (ucs_unlikely(vec_len == 0))
		return;

	if (is_peers) {
		dst_vec = allreduce_super_request->peer_result_vector;
		src_vecs = allreduce_super_request->recv_vectors;
		nb_vecs = allreduce_super_request->recv_vector_iter;
	} else {
		dst_vec = allreduce_super_request->result_vector;
		src_vecs = allreduce_super_request->clients_recv_vectors;
		nb_vecs = allreduce_config.num_clients;
		/* If the result vector was taken from a client */
		if (allreduce_super_request->result_vector_owner)
			--nb_vecs;
	}

	/* Threshold is set at the intersection point of the complexity functions, also we want at least 4 vecs. */
	const size_t x = (nb_vecs <= 2 * gpu_info.max_thrds_per_blk) ? nb_vecs : 2 * gpu_info.max_thrds_per_blk;
	const size_t y = x * ((MIN_VEC_LEN_PER_THREAD * gpu_info.max_grid_dim_x * gpu_info.max_grid_dim_y) / vec_len);

	if (nb_vecs > 4 && ucs_likely(y >= 2 * __builtin_clzl(x)))
		reduce_all_for_many_vecs(dst_vec, src_vecs, nb_vecs, vec_len, *allreduce_super_request->stream);
	else {
		struct vectors src = {};
		src.arr = src_vecs;
		src.n = nb_vecs;
		gpu_reduce(dst_vec, src, vec_len, *allreduce_super_request->stream);
	}
}

void
set_cuda_globals(void)
{
	int dev_id;
	int streams_overlap_enabled;

	/* Prevent any useless profiling */
	hipProfilerStop();

	/* Get GPU info */
	CUDA_ASSERT(hipGetDevice(&dev_id));
	gpu_info.dev_id = dev_id;
	CUDA_ASSERT(hipDeviceGetAttribute(&gpu_info.max_thrds_per_blk, hipDeviceAttributeMaxThreadsPerBlock, dev_id));
	CUDA_ASSERT(hipDeviceGetAttribute(&gpu_info.max_grid_dim_x, hipDeviceAttributeMaxGridDimX, dev_id));
	CUDA_ASSERT(hipDeviceGetAttribute(&gpu_info.max_grid_dim_y, hipDeviceAttributeMaxGridDimY, dev_id));
	CUDA_ASSERT(hipDeviceGetAttribute(&streams_overlap_enabled, hipDeviceAttributeAsyncEngineCount, dev_id));
	CUDA_ASSERT(hipDeviceGetAttribute(&gpu_info.warp_size, hipDeviceAttributeWarpSize, dev_id));

	if (!streams_overlap_enabled)
		DOCA_LOG_WARN("GPU overlapping is disabled - please enable it for better performance.");

	/* Allocate constant GPU memory */
	CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(datatype), &allreduce_config.datatype, sizeof(allreduce_config.datatype)));
	CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(operation), &allreduce_config.operation, sizeof(allreduce_config.operation)));
}

void
allreduce_reduce(struct allreduce_super_request *allreduce_super_request, void *src_vec, bool is_peer)
{
	void *dst_vec = is_peer ? allreduce_super_request->peer_result_vector : allreduce_super_request->result_vector;
	size_t dst_vec_len = allreduce_super_request->result_vector_size;
	hipStream_t stream = *allreduce_super_request->stream;
	struct vectors src = {};

	src.vec = src_vec;
	src.n = 1;

	if (ucs_unlikely(dst_vec_len == 0))
		return;

	gpu_reduce(dst_vec, src, dst_vec_len, stream);
}
