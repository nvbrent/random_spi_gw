#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES, ALL RIGHTS RESERVED.
 *
 * This software product is a proprietary product of NVIDIA CORPORATION &
 * AFFILIATES (the "Company") and all right, title, and interest in and to the
 * software product, including all associated intellectual property rights, are
 * and shall remain exclusively with the Company.
 *
 * This software product is governed by the End User License Agreement
 * provided with the software product.
 *
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <gpu_init.h>	/* Should put it before DPDK header includes, it contains a define to disable DPDK warnings */

#include <rte_ethdev.h>
#include <rte_gpudev.h>

extern "C" {
void workload_launch_gpu_processing(struct rte_gpu_comm_list *comm_list, hipStream_t c_stream, char **queries);
}

#define DNS_FLAGS_SIZE 12	/* DNS packet payload starts with flags and details, the size is 12B */

/*
 * Calculate IPV4 header length
 *
 * @ipv4_hdr [in]: packet IPV4 header
 * @return: packet IPV4 header length
 */
__device__ __forceinline__ uint8_t
gpu_ipv4_hdr_len(const struct rte_ipv4_hdr *ipv4_hdr)
{
	return (uint8_t)((ipv4_hdr->version_ihl & RTE_IPV4_HDR_IHL_MASK) * RTE_IPV4_IHL_MULTIPLIER);
};

/*
 * CUDA kernel function to inspect the packets burst and extract the DNS queries
 *
 * @comm_list [in]: array of communication objects, holds the bursted packets context
 * @queries [out]: array of DNS queries
 */
__global__ void
gpu_dns_workload(struct rte_gpu_comm_list *comm_list, char **queries)
{
	/* thread_ID, each thread works on packet with index=thread_ID */
	int idx = threadIdx.x;

	/* Ethernet layer header size, skip it to reach the L3 header */
	const int l2_len = RTE_ETHER_HDR_LEN;
	const struct rte_ether_hdr *eth_hdr = (const struct rte_ether_hdr *) comm_list->pkt_list[idx].addr;
	const uint8_t *l3_hdr = (const uint8_t *)eth_hdr + l2_len;
	const struct rte_ipv4_hdr *hdr = (const struct rte_ipv4_hdr *) l3_hdr;

	/* Calculate L3 header size, skip it to reach L4 headers */
	uint8_t ip_hdr_len = gpu_ipv4_hdr_len(hdr);
	const uint8_t *l4_hdr = (const uint8_t *) (l3_hdr + ip_hdr_len);

	/* Calculate DNS query offset */
	int offset = l4_hdr - (const uint8_t *) comm_list->pkt_list[idx].addr;
	offset += sizeof(struct rte_udp_hdr); /* UDP Header size = 8B */
	offset += DNS_FLAGS_SIZE; /* Skip DNS flags */

	/* Store the address of DNS query */
	queries[idx] = (char *)(comm_list->pkt_list[idx].addr + offset);

	__syncthreads(); /* Wait all threads to reach this point */
	if (idx == 0) {
		/* Notify that GPU workload is done */
		RTE_GPU_VOLATILE(*(comm_list->status_d)) = RTE_GPU_COMM_LIST_DONE;
	}
}

void
workload_launch_gpu_processing(struct rte_gpu_comm_list *comm_list, hipStream_t c_stream, char **queries)
{
	/* Create CUDA kernel to start GPU workload */
#ifdef DOCA_LOGGING_ALLOW_DLOG
	printf("CUDA kernel launch for extracting DNS queries\n");
#endif
	gpu_dns_workload<<<1, comm_list->num_pkts, 0, c_stream>>>(comm_list, queries);
}
